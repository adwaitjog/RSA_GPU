#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>

#include <openssl/rsa.h>
#include <openssl/err.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>

#include <sys/socket.h>
#include <netinet/tcp.h>
#include <arpa/inet.h>
#include <unistd.h>

#include "common.hh"
#include "rsa_cpu.hh"
#include "rsa_gpu.hh"

#define checkCurandErrors(x) do { hiprandStatus_t status= (x);\
	if (status != HIPRAND_STATUS_SUCCESS) { \
	printf("Error %d at %s:%d\n", status, __FILE__, __LINE__); \
	exit(EXIT_FAILURE);}} while(0)

__global__ void gpu_modexp(int msg_num, int msg_size, WORD *input, WORD *output);

int main(int argc, char *argv[]) {
	srand(time(NULL));
	int key_bits = 512;
	const char *key_file = "private_key.pem";
	/* Generate key file if not exits */
	FILE *file =fopen(key_file, "r");
	if(file!=NULL) {
		fclose(file);
		printf("Key file %s exists.\n", key_file);
	}
	else {
		Gen_RSA_key(key_bits, key_file);
		printf("Key file %s generated.\n", key_file);
	}
	/* Read key file into memory */
	RSA *rsa = Read_key_file(key_file);
	assert(rsa != NULL);
	key_bits = RSA_size(rsa)*8;
	int key_bytes = key_bits/8;
	int msg_bytes = key_bytes;
	int msg_size = key_bytes/sizeof(WORD);
	printf("key bits: %d\n", key_bits);
	/* store key in to WORD array */
	WORD e[msg_size];
	WORD d[msg_size];
	WORD n[msg_size];
	WORD np[msg_size];
	WORD r_sqr[msg_size];

	WORD pd[msg_size/2];
	WORD qd[msg_size/2];
	WORD p[msg_size/2];
	WORD q[msg_size/2];
	WORD pp[msg_size/2];
	WORD qp[msg_size/2];
	WORD pr_sqr[msg_size/2];
	WORD qr_sqr[msg_size/2];
	WORD iqmp[msg_size/2];

	key_setup(rsa, e, d, n ,np, r_sqr,
		pd, qd, p, q, pp, qp, pr_sqr, qr_sqr, iqmp);
	int mem_bytes = key_bytes * 56 * 1024;
	gpu_setup(msg_size, d, n, np, r_sqr, mem_bytes,
		pd, qd, p, q, pp, qp, pr_sqr, qr_sqr, iqmp, rsa);

	int msg_num = 56;
	int trace_num = 1;
	if (argc >= 2) msg_num = atoi(argv[1]);
	if (argc >= 3) trace_num = atoi(argv[2]);
	printf("msg_num: %d, trace_num: %d.\n", msg_num, trace_num);
	mem_bytes = msg_bytes*msg_num*trace_num;
	/* Random number Gen setup */
	WORD *msges;
	checkCudaErrors(hipMalloc(&msges, mem_bytes));
	hiprandGenerator_t gen;
	checkCurandErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
	checkCurandErrors(hiprandSetPseudoRandomGeneratorSeed(gen, 0));
	checkCurandErrors(hiprandGenerate(gen, (unsigned int *)msges, 
		mem_bytes / sizeof(unsigned int)));
	checkCurandErrors(hiprandDestroyGenerator(gen));
	checkCudaErrors(hipDeviceSynchronize());

	/* TCP/IP setting */
	int listenfd = 0, connfd = 0;
	struct sockaddr_in serv_addr;
	char sendBuff[1024], recvBuff[1024];
	int socket_err;
	int flag = 1;
	listenfd = socket(AF_INET, SOCK_STREAM, 0);
	setsockopt(listenfd, IPPROTO_TCP, TCP_NODELAY, (char *) &flag, sizeof(int));
	setsockopt(listenfd, SOL_SOCKET, SO_REUSEADDR, (char *) &flag, sizeof(int));
	memset(&serv_addr, 0, sizeof(serv_addr));
	memset(sendBuff, 0, sizeof(sendBuff));
	memset(recvBuff, 0, sizeof(recvBuff));
	serv_addr.sin_family = AF_INET;
	serv_addr.sin_addr.s_addr = htonl(INADDR_ANY);
	serv_addr.sin_port = htons(5000);
	socket_err = bind(listenfd, (struct sockaddr*) &serv_addr, sizeof(serv_addr));
	if (socket_err < 0) {
		printf("Bind err: %d\n", errno);
		exit(EXIT_FAILURE);
	}
	listen(listenfd, 10);
	connfd = accept(listenfd, (struct sockaddr*) NULL, NULL);
	if (connfd <0) {
		printf("tcp accept error\n");
		exit(EXIT_FAILURE);
	}

	uint64_t start, end;
	int grid_size = (msg_num * msg_size + BLK_SIZE - 1) / BLK_SIZE;
	printf("block size: %d, grid size: %d.\n", BLK_SIZE, grid_size);
	start = get_usec();
	int pre_exit = 0;
	for (int i = 0; i < trace_num; i++) {
		int n = recv(connfd, recvBuff, sizeof(recvBuff)-1, 0);
		if (n > 0) {
			recvBuff[n] = 0;
			printf("received command: %s\n", recvBuff);
		}
		if ((strcmp(recvBuff, "exit")) == 0) {
			printf("exit\n");
			send(connfd, "exit\r\n", 6, 0);
			pre_exit = 1;
			break;
		}
		printf("trace %d\n", i);
		sprintf(sendBuff, "trace %d\r\n", i);
		send(connfd, sendBuff, strlen(sendBuff), 0);
		gpu_modexp<<<grid_size, BLK_SIZE>>>(msg_num, msg_size, msges + i * msg_num * msg_size,
			msges + i * msg_num * msg_size);
		checkCudaErrors(hipDeviceSynchronize());
	}
	if (pre_exit == 0) {
		printf("finished\n");
		send(connfd, "finished\r\n", 10, 0);
	}
	end = get_usec();
	printf("GPU multiple msg test okay, time(us): %lu.\n", end-start);

	/* Clear memory */
	close(connfd);
	close(listenfd);
	RSA_free(rsa);
	checkCudaErrors(hipFree(msges));
	gpu_reset();
	return 0;
}
