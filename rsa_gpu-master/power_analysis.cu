#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>

#include <openssl/rsa.h>
#include <openssl/err.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>

#include "common.hh"
#include "rsa_cpu.hh"
#include "rsa_gpu.hh"
#include "tanc.hh"

#define checkCurandErrors(x) do { hiprandStatus_t status= (x);\
	if (status != HIPRAND_STATUS_SUCCESS) { \
	printf("Error %d at %s:%d\n", status, __FILE__, __LINE__); \
	exit(EXIT_FAILURE);}} while(0)

/* msg copy: every trace will have one msg but msg_num replicas*/
__global__ void gpu_memcpy(int msg_num, int msg_size, int trace_num, WORD *msges) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= trace_num) return;
	int idx = tid * msg_num * msg_size;
	for (int i = 1; i < msg_num; i++) {
		for (int j = 0; j < msg_size; j++) {
			msges[idx + i * msg_size + j] = msges[idx + j];
		}
	}
}

/* squeeze out the unused or redundant msges*/
__global__ void gpu_memdecimate(int msg_num, int msg_size, int trace_num, WORD *msges) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= trace_num) return;
	int idx = tid * msg_num * msg_size;
	for (int i = 0; i < msg_size; i++) {
		msges[tid * msg_size + i] = msges[idx + i];
	}
}

int main(int argc, char *argv[]) {
	if (argc < 4) {
		printf("Usage: %s msg_num trace_num power_date_file\n", argv[0]);
		exit(1);
	}
	/* disable in/output buffering */
	setvbuf(stdin, NULL, _IONBF, 0);
	setvbuf(stdout, NULL, _IONBF, 0);

	srand(time(NULL));
	int key_bits = 512;
	const char *key_file = "private_key.pem";
	/* Generate key file if not exits */
	FILE *file =fopen(key_file, "r");
	if(file!=NULL) {
		fclose(file);
		printf("Key file %s exists.\n", key_file);
	}
	else {
		Gen_RSA_key(key_bits, key_file);
		printf("Key file %s generated.\n", key_file);
	}
	/* Read key file into memory */
	RSA *rsa = Read_key_file(key_file);
	assert(rsa != NULL);
	key_bits = RSA_size(rsa)*8;
	int key_bytes = key_bits/8;
	int msg_bytes = key_bytes;
	int msg_size = key_bytes/sizeof(WORD);
	printf("key bits: %d\n", key_bits);
	/* store key in to WORD array */
	WORD e[msg_size];
	WORD d[msg_size];
	WORD n[msg_size];
	WORD np[msg_size];
	WORD r_sqr[msg_size];

	WORD pd[msg_size/2];
	WORD qd[msg_size/2];
	WORD p[msg_size/2];
	WORD q[msg_size/2];
	WORD pp[msg_size/2];
	WORD qp[msg_size/2];
	WORD pr_sqr[msg_size/2];
	WORD qr_sqr[msg_size/2];
	WORD iqmp[msg_size/2];

	key_setup(rsa, e, d, n ,np, r_sqr,
		pd, qd, p, q, pp, qp, pr_sqr, qr_sqr, iqmp);
	int mem_bytes = key_bytes * 56 * 1024;
	gpu_setup(msg_size, d, n, np, r_sqr, mem_bytes,
		pd, qd, p, q, pp, qp, pr_sqr, qr_sqr, iqmp, rsa);

	int msg_num = 56;
	int trace_num = 1;
	if (argc >= 2) msg_num = atoi(argv[1]);
	if (argc >= 3) trace_num = atoi(argv[2]);
	printf("msg_num: %d, trace_num: %d, file: %s.\n", msg_num, trace_num, argv[3]);
	mem_bytes = msg_bytes*msg_num*trace_num;

	/* Random number Gen setup */
	WORD *msges;
	checkCudaErrors(hipMalloc(&msges, mem_bytes));
	hiprandGenerator_t gen;
	checkCurandErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
	checkCurandErrors(hiprandSetPseudoRandomGeneratorSeed(gen, 0));
	checkCurandErrors(hiprandGenerate(gen, (unsigned int *)msges, 
		mem_bytes / sizeof(unsigned int)));
	checkCurandErrors(hiprandDestroyGenerator(gen));
	checkCudaErrors(hipDeviceSynchronize());
	int grid_size = (trace_num + 256 -1) / 256;
	gpu_memdecimate<<<grid_size, 256>>>(msg_num, msg_size, trace_num, msges);

	/* load power data */
	char *file_name = argv[3];
	FILE *dfile = fopen(file_name, "rb");
	double *power_h = (double *) malloc(sizeof(double) * trace_num);
	fread(power_h, sizeof(double), trace_num, dfile);

	/* attack */
	WORD *msges0, *msges1, *msges_mont;
	checkCudaErrors(hipMalloc(&msges0, msg_bytes*trace_num));
	checkCudaErrors(hipMalloc(&msges1, msg_bytes*trace_num));
	checkCudaErrors(hipMalloc(&msges_mont, msg_bytes*trace_num));
	/* pre-processing */
	grid_size = (msg_size * trace_num + BLK_SIZE -1) / BLK_SIZE;
	gpu_preprocessing<<<grid_size, BLK_SIZE>>>
		(msg_num * trace_num, msg_size, msges, msges0, msges_mont);
	/* print msges */
	/*WORD *msg_h = (WORD *)malloc(10 * msg_bytes);
	checkCudaErrors(hipMemcpy(msg_h, msges_mont, 10*msg_bytes, hipMemcpyDeviceToHost));
	for (int i = 0; i < 10; i++) {
		for (int j = 0; j < msg_size; j++) {
			printf("%016lx", msg_h[i*msg_size+j]);
		}
		printf("\n");
	}
	free(msg_h);*/
	dt_attack(msg_size, 1, trace_num, 3, 510/3, power_h, d, msges0, msges1, msges_mont);

	/* Clear memory */
	free(power_h);
	checkCudaErrors(hipFree(msges0));
	checkCudaErrors(hipFree(msges1));
	checkCudaErrors(hipFree(msges_mont));
	RSA_free(rsa);
	checkCudaErrors(hipFree(msges));
	gpu_reset();
	return 0;
}
